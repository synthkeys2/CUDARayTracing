#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

//#include <stdio.h>
#include "RayTracer.h"
//#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"


__global__ void RayTracer(uchar4* dest, const int imageW, const int imageH, float4 cameraLocation, float4 cameraUp, float4 cameraForward, float4 cameraRight, float nearPlaneDistance, float2 viewSize)
{
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;

	// Compute the location in the dest array that will be written to
	const int pixel = imageW * iy + ix;

	// Compute the center of the near plane. All rays will be computed as an offset from this point
	const float4 lookAt = cameraLocation + cameraForward * nearPlaneDistance;

	const float4 rayMidPoint = lookAt + cameraRight * ((float(ix) / float(imageW) - 0.5f) * viewSize.x) + cameraUp * ((1 - (float(iy) / float(imageH)) - 0.5f) * viewSize.y); 
	const float4 ray = normalize(rayMidPoint - cameraLocation);

	const float4 sphereCenter = make_float4(0, 0, 20, 1);
	const float radius = 10.0f;

	const float4 rayOriginMinusSphereCenter = cameraLocation - sphereCenter;

	const float A = dot(ray, ray);
	const float B = 2 * dot(rayOriginMinusSphereCenter, ray);
	const float C = dot(rayOriginMinusSphereCenter, rayOriginMinusSphereCenter) - radius * radius;

	const float disc = B * B - 4 * A * C;

	float t = -1.0f;

	if(disc >= 0)
	{
		const float discSqrt = sqrtf(disc);
		float q;
		
		if(B < 0)
		{
			q = (-B - discSqrt) / 2.0f;
		}
		else
		{
			q = (-B + discSqrt) / 2.0f;
		}

		float t0 = q / A;
		float t1 = C / q;

		if(t0 > t1)
		{
			float temp = t0;
			t0 = t1;
			t1 = temp;
		}

		if(t1 < 0)
		{
			
		}
		else if(t0 < 0)
		{
			t = t1;
		}
		else
		{
			t = t0;
		}
	}

	if(t < 0)
	{
		dest[pixel] = make_uchar4(255, 255, 255, 255);
	}
	else
	{
		dest[pixel] = make_uchar4(100, 0, 100, 255);
	}
}


void RunRayTracer(uchar4* dest, const int imageW, const int imageH, const int xThreadsPerBlock, const float4 a_vCameraLocation, const float a_fNearPlaneDistance)
{
	dim3 numThreads(20, 20);
	dim3 numBlocks(64, 36);

	float4 cameraUp, cameraForward, cameraRight;
	float2 viewSize;

	cameraUp = make_float4(0, 1, 0, 0);
	cameraForward = make_float4(0, 0, 1, 0);
	cameraRight = make_float4(1, 0, 0, 0);
	viewSize = make_float2(imageW, imageH);

	RayTracer<<<numBlocks, numThreads>>>(dest, imageW, imageH, a_vCameraLocation, cameraUp, cameraForward, cameraRight, a_fNearPlaneDistance, viewSize);
}