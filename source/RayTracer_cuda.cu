#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

//#include <stdio.h>
#include "RayTracer.h"
//#include "hip/hip_runtime_api.h"


__global__ void RayTracer(uchar4* dest, const int imageW, const int imageH)
{
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;

	const int pixel = imageW * iy + ix;

	dest[pixel].y = pixel % 255;
}


void RunRayTracer(uchar4* dest, const int imageW, const int imageH, const int xThreadsPerBlock)
{
	dim3 numThreads(20, 20);
	dim3 numBlocks(64, 36);

	RayTracer<<<numBlocks, numThreads>>>(dest, imageW, imageH);
}